
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Forward_gpu(
    const vector<Blob<Dtype,Mtype>*>& bottom, const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();


    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      MemoryHandler::mallocGPU(&workspaceData, workspace_fwd_sizes_[i]);
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspaceData, workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      MemoryHandler::freeGPU(workspaceData);
      workspaceData = NULL;
      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor_v3(Caffe::cudnn_handle(),
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }

    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
  }
}


template <typename Dtype, typename Mtype>
void CuDNNConvolutionLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;

  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_set<Dtype,Mtype>(this->blobs_[0]->count(), Mtype(0), weight_diff);
  }
  Dtype* bias_diff = NULL;

  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_set<Dtype,Mtype>(this->blobs_[1]->count(), Mtype(0), bias_diff);
  }

  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();

    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(),
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        MemoryHandler::mallocGPU(&workspaceData, workspace_bwd_filter_sizes_[i]);
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter_v3(
              Caffe::cudnn_handle(),
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspaceData, workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
        MemoryHandler::freeGPU(workspaceData);
        workspaceData = NULL;
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        MemoryHandler::mallocGPU(&workspaceData, workspace_bwd_data_sizes_[i]);
        CUDNN_CHECK(cudnnConvolutionBackwardData_v3(
              Caffe::cudnn_handle(),
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspaceData, workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
        MemoryHandler::freeGPU(workspaceData);
        workspaceData = NULL;
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
